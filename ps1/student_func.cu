#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__ void rgba_to_greyscale(const uchar4* const rgbaImage,
                                  unsigned char* const greyImage,
                                  int numRows, int numCols)
{
    //TODO
    //Fill in the kernel to convert from color to greyscale
    //the mapping from components of a uchar4 to RGBA is:
    // .x -> R ; .y -> G ; .z -> B ; .w -> A
    //
    //The output (greyImage) at each pixel should be the result of
    //applying the formula: output = .299f * R + .587f * G + .114f * B;
    //Note: We will be ignoring the alpha channel for this conversion

    //First create a mapping from the 2D block and grid locations
    //to an absolute 2D location in the image, then use that to
    //calculate a 1D offset

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= numRows || y >= numCols)
    {
        return;
    }

    int index = (x * numCols) + y;
    uchar4 pixel = rgbaImage[index];

    char greyPixel = (.299f * pixel.x) + (.587f * pixel.y) + (.114f * pixel.z);
    greyImage[index] = greyPixel;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
    // Create a block of threads 32*32 (1024 is max threads per block)
    // Then make enough blocks to have one thread per pixel
    int size = 32;
    int gridWidth = numRows/size + 1;
    int gridHeight = numCols/size + 1;

    const dim3 blockSize(size, size, 1);
    const dim3 gridSize( gridWidth, gridHeight, 1);
    rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
